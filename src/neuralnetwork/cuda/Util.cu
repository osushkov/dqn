
#include "Util.hpp"
#include <iostream>
#include <cassert>

using namespace neuralnetwork::cuda;

void util::OutputError(hipError_t code, const char *file, int line) {
  if (code != hipSuccess) {
    std::cerr << "GPU error: " << hipGetErrorString(code) << " "
        << file << "(" << line << ")" << std::endl;
    exit(code);
  }
}

void *util::AllocPushBuffer(size_t bufSize) {
  void* result = nullptr;

  hipError_t err = hipHostAlloc(&result, bufSize, hipHostMallocWriteCombined);
  CheckError(err);
  assert(result != nullptr);

  return result;
}

void util::FreePushBuffer(void *buf) {
  assert(buf != nullptr);
  hipError_t err = hipHostFree(buf);
  CheckError(err);
}

LayerWeights util::NewLayerWeights(unsigned inputSize, unsigned layerSize) {
  assert(inputSize > 0 && layerSize > 0);

  LayerWeights result;
  result.inputSize = inputSize;
  result.layerSize = layerSize;

  size_t width = inputSize * sizeof(float);
  size_t height = layerSize;

  // hipError_t err = hipMalloc(&result.weights, width * height);
  // result.pitch = width;
  hipError_t err = hipMallocPitch(&(result.weights), &(result.pitch), width, height);
  CheckError(err);

  return result;
}

void util::DeleteLayerWeights(LayerWeights &lw) {
  hipError_t err = hipFree(lw.weights);
  CheckError(err);
  lw.weights = nullptr;
}

SamplesBatch util::NewSamplesBatch(unsigned maxBatchSize, unsigned inputDim,
                                   unsigned targetOutputDim) {
  assert(maxBatchSize > 0 && inputDim > 0 && targetOutputDim > 0);

  SamplesBatch result;
  result.maxBatchSize = maxBatchSize;
  result.batchSize = 0;
  result.inputDim = inputDim;
  result.targetOutputDim = targetOutputDim;

  size_t width = inputDim * sizeof(float);
  size_t height = maxBatchSize;

  hipError_t err = hipMallocPitch(&result.input, &result.ipitch, width, height);
  CheckError(err);

  width = targetOutputDim * sizeof(float);
  err = hipMallocPitch(&result.targetOutput, &result.opitch, width, height);
  CheckError(err);

  return result;
}

void util::DeleteSamplesBatch(SamplesBatch &sb) {
  hipError_t err = hipFree(sb.input);
  CheckError(err);
  sb.input = nullptr;

  err = hipFree(sb.targetOutput);
  CheckError(err);
  sb.targetOutput = nullptr;
}

LayerBatchOutputs util::NewLayerBatchOutputs(unsigned maxBatchSize, unsigned layerSize) {
  assert(maxBatchSize > 0 && layerSize > 0);

  LayerBatchOutputs result;
  result.maxBatchSize = maxBatchSize;
  result.batchSize = 0;
  result.layerSize = layerSize;

  size_t width = layerSize * sizeof(float);
  size_t height = maxBatchSize;

  hipError_t err = hipMallocPitch(&result.output, &result.opitch, width, height);
  CheckError(err);

  err = hipMallocPitch(&result.derivative, &result.dpitch, width, height);
  CheckError(err);

  return result;
}

void util::DeleteLayerBatchOutputs(LayerBatchOutputs &lbo) {
  hipError_t err = hipFree(lbo.output);
  CheckError(err);
  lbo.output = nullptr;

  err = hipFree(lbo.derivative);
  CheckError(err);
  lbo.derivative = nullptr;
}

LayerBatchDeltas util::NewLayerBatchDeltas(unsigned maxBatchSize, unsigned layerSize) {
  assert(maxBatchSize > 0 && layerSize > 0);

  LayerBatchDeltas result;
  result.maxBatchSize = maxBatchSize;
  result.batchSize = 0;
  result.layerSize = layerSize;

  size_t width = layerSize * sizeof(float);
  size_t height = maxBatchSize;

  hipError_t err = hipMallocPitch(&result.delta, &result.pitch, width, height);
  CheckError(err);

  return result;
}

void util::DeleteLayerBatchDeltas(LayerBatchDeltas &lbd) {
  hipError_t err = hipFree(lbd.delta);
  CheckError(err);
  lbd.delta = nullptr;
}
