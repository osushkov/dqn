#include "hip/hip_runtime.h"
#include "TargetValuesKernel.hpp"
#include <hip/hip_runtime.h>

using namespace neuralnetwork::cuda;

__global__ void targetValuesKernel(LayerBatchOutputs outputs, SamplesBatch samplesBatch) {
  assert(gridDim.x == outputs.batchSize);
  const unsigned batchIndex = blockIdx.x;

  if (samplesBatch.isTerminal[batchIndex]) {
    samplesBatch.targetOutput[batchIndex] = samplesBatch.rewards[batchIndex];
  } else {
    float maxVal = *(outputs.OutputElem(batchIndex, 0));
    for (unsigned i = 1; i < outputs.layerSize - 1; i++) {
      maxVal = fmaxf(maxVal, *(outputs.OutputElem(batchIndex, i)));
    }

    samplesBatch.targetOutput[batchIndex] = samplesBatch.rewards[batchIndex] +
        samplesBatch.futureRewardDiscount * maxVal;
  }
}

void TargetValuesKernel::Apply(const LayerBatchOutputs &lastLayer, const SamplesBatch &samplesBatch,
                               hipStream_t stream) {
  int tpb = 1;
  int bpg = lastLayer.batchSize;
  targetValuesKernel<<<bpg, tpb, 0, stream>>>(lastLayer, samplesBatch);
}
