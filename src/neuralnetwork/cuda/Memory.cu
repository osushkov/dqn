
#include "Memory.hpp"
#include "Util.hpp"
#include <hip/hip_runtime.h>

using namespace neuralnetwork::cuda;

void *memory::AllocPushBuffer(size_t bufSize) {
  void* result = nullptr;

  hipError_t err = hipHostAlloc(&result, bufSize, hipHostMallocWriteCombined);
  CheckError(err);
  assert(result != nullptr);

  return result;
}

void memory::FreePushBuffer(void *buf) {
  assert(buf != nullptr);
  hipError_t err = hipHostFree(buf);
  CheckError(err);
}
