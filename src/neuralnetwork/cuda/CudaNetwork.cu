#include "hip/hip_runtime.h"

#include "CudaNetwork.hpp"
#include "Util.hpp"
#include "SoftmaxKernel.hpp"
#include "ForwardPassKernel.hpp"
#include "TransposeKernel.hpp"
#include "BackwardDeltaKernel.hpp"
#include "GradientKernel.hpp"
#include "TargetValuesKernel.hpp"
#include "Constants.hpp"
#include "Random.hpp"

#include <cassert>
#include <cmath>
#include <vector>
#include <iostream>
#include <cstdio>
#include <mutex>

#include <hiprand.h>
#include <hip/hip_runtime.h>

using namespace neuralnetwork;
using namespace neuralnetwork::cuda;
using namespace std;

// ADAM trainer parameters
static constexpr float adamBeta1 = 0.9f;
static constexpr float adamBeta2 = 0.999f;
static constexpr float adamEpsilon = 10e-8;
static constexpr float adamLearnRate = 0.001f;

static Random rnd;
static std::once_flag stateFlag;

static constexpr unsigned NUM_BUFFERS = 2;

static void initialiseSharedState(void) {
  std::call_once(stateFlag, [](){
    rnd = Random::Create(2048, 1337);
  });
}

__global__ void initialiseLayerWeights(LayerWeights layer, const float initRange, Random rnd) {
  const unsigned row = blockDim.y * blockIdx.y + threadIdx.y;
  const unsigned col = blockDim.x * blockIdx.x + threadIdx.x;

  if (row >= layer.layerSize || col >= layer.inputSize) {
    return;
  }

  float *out = layer.Elem(row, col);
  *out = initRange * (rnd.SampleUniform(col + row * layer.inputSize) * 2.0f - 1.0f);
}

__global__ void initialiseLayerOutputs(LayerBatchOutputs outputs) {
  const unsigned id = blockDim.x * blockIdx.x + threadIdx.x;
  if (id >= outputs.maxBatchSize) {
    return;
  }

  *(outputs.OutputElem(id, outputs.layerSize - 1)) = 1.0f;
}

__global__ void initialiseAdamWeights(LayerWeights momentum, LayerWeights rms) {
  assert(momentum.inputSize == rms.inputSize);
  assert(momentum.layerSize == rms.layerSize);

  const unsigned row = blockDim.y * blockIdx.y + threadIdx.y;
  const unsigned col = blockDim.x * blockIdx.x + threadIdx.x;

  if (row >= rms.layerSize || col >= rms.inputSize) {
    return;
  }

  *momentum.Elem(row, col) = 0.0f;
  *rms.Elem(row, col) = 0.0f;
}

__global__ void lastLayerDeltasKernel(LayerBatchOutputs networkOutput, SamplesBatch samples,
                                      LayerBatchDeltas out) {
  const unsigned row = blockDim.y * blockIdx.y + threadIdx.y;
  const unsigned col = blockDim.x * blockIdx.x + threadIdx.x;

  if (row >= out.batchSize || col >= out.layerSize) {
    return;
  }

  float delta = 0.0f;
  if (col == samples.actionIndex[row]) {
    float out = *networkOutput.OutputElem(row, col);
    // delta = out * (1.0f - out) * (out - samples.targetOutput[row]);
    delta = (out - samples.targetOutput[row]);
  }

  *out.Elem(row, col) = delta;
}

__global__ void updateMomentumAndRMS(LayerWeights gradient, LayerWeights momentum, LayerWeights rms,
                                      const float beta1, const float beta2) {
  const unsigned row = blockDim.y * blockIdx.y + threadIdx.y;
  const unsigned col = blockDim.x * blockIdx.x + threadIdx.x;

  if (row >= gradient.layerSize || col >= gradient.inputSize) {
    return;
  }

  float g = *gradient.Elem(row, col);
  float m = *momentum.Elem(row, col);
  float r = *rms.Elem(row, col);

  *momentum.Elem(row, col) = m * beta1 + g * (1.0f - beta1);
  *rms.Elem(row, col) = r * beta2 + g * g * (1.0f - beta2);
}

__global__ void updateWeightsWithAdam(LayerWeights weights, LayerWeights momentum, LayerWeights rms,
                                      const float beta1, const float beta2,
                                      const float lr, const float epsilon) {

  const unsigned row = blockDim.y * blockIdx.y + threadIdx.y;
  const unsigned col = blockDim.x * blockIdx.x + threadIdx.x;

  if (row >= rms.layerSize || col >= rms.inputSize) {
    return;
  }

  float mc = *momentum.Elem(row, col) / (1.0f - beta1);
  float rc = *rms.Elem(row, col) / (1.0f - beta2);

  *weights.Elem(row, col) -= lr * mc / sqrtf(rc + epsilon);
}

struct CudaNetwork::CudaNetworkImpl {
  NetworkSpec networkSpec;
  vector<LayerWeights> d_layerWeights;
  vector<LayerWeights> d_targetLayerWeights;
  vector<LayerWeights> d_layerGradients;
  vector<LayerBatchOutputs> d_layerOutputs;
  vector<LayerBatchDeltas> d_layerDeltas;
  LayerWeights d_transposeScratch;

  // TODO: this stuff should go into a separate file. Trainer code/variables should be
  // separate from network code.
  vector<LayerWeights> d_adamMomentum;
  vector<LayerWeights> d_adamRMS;

  unsigned curStream = 0;
  unsigned otherStream = 1;
  vector<SamplesBatch> d_samplesBatch;
  vector<hipStream_t> computeStream;

  CudaNetworkImpl(const NetworkSpec &spec) : networkSpec(spec) {
    assert(networkSpec.hiddenActivation != LayerActivation::SOFTMAX);
    initialiseSharedState();

    computeStream.resize(NUM_BUFFERS);
    for (unsigned i = 0; i < NUM_BUFFERS; i++) {
      hipStreamCreate(&computeStream[i]);
    }

    allocDeviceMemory();
    initialiseWeights();
    initialiseOutputs();
    initialiseADAM();
  }

  ~CudaNetworkImpl() {
    for (auto& lw : d_layerWeights) { util::DeleteLayerWeights(lw); }
    for (auto& lw : d_targetLayerWeights) { util::DeleteLayerWeights(lw); }
    for (auto& lg : d_layerGradients) { util::DeleteLayerWeights(lg); }
    for (auto& lo : d_layerOutputs) { util::DeleteLayerBatchOutputs(lo); }
    for (auto& ld : d_layerDeltas) { util::DeleteLayerBatchDeltas(ld); }
    for (auto& am : d_adamMomentum) { util::DeleteLayerWeights(am); }
    for (auto& am : d_adamRMS) { util::DeleteLayerWeights(am); }
    for (auto& sb : d_samplesBatch) { util::DeleteSamplesBatch(sb); }
    util::DeleteLayerWeights(d_transposeScratch);
  }

  void SetWeights(const std::vector<math::MatrixView> &weights) {
    assert(d_layerWeights.size() == weights.size());

    for (unsigned i = 0; i < weights.size(); i++) {
      assert(weights[i].rows == d_layerWeights[i].layerSize);
      assert(weights[i].cols == d_layerWeights[i].inputSize);

      hipError_t err = hipMemcpy2D(
          d_layerWeights[i].weights, d_layerWeights[i].pitch,
          weights[i].data, weights[i].cols * sizeof(float),
          weights[i].cols * sizeof(float), weights[i].rows,
          hipMemcpyHostToDevice);

      CheckError(err);

      err = hipMemcpy2D(
          d_targetLayerWeights[i].weights, d_targetLayerWeights[i].pitch,
          weights[i].data, weights[i].cols * sizeof(float),
          weights[i].cols * sizeof(float), weights[i].rows,
          hipMemcpyHostToDevice);

      CheckError(err);
    }
  }

  void GetWeights(std::vector<math::MatrixView> &outWeights) {
    assert(outWeights.size() == d_targetLayerWeights.size());

    for (unsigned i = 0; i < outWeights.size(); i++) {
      assert(outWeights[i].rows == d_targetLayerWeights[i].layerSize);
      assert(outWeights[i].cols == d_targetLayerWeights[i].inputSize);

      hipError_t err = hipMemcpy2D(
          outWeights[i].data, outWeights[i].cols * sizeof(float), // dst
          d_targetLayerWeights[i].weights, d_targetLayerWeights[i].pitch, // src
          outWeights[i].cols * sizeof(float), outWeights[i].rows, // width, height
          hipMemcpyDeviceToHost);

      CheckError(err);
    }
  }

  void UpdateTarget(void) {
    updateTargetWeights();
  }

  void Train(const QBatch &qbatch) {

    // for (unsigned i = 0; i < targetOutputs.size(); i++) {
    //   std::cout << targetOutputIndices[i] << " : " << targetOutputs[i] << std::endl;
    // }
    // std::cout << std::endl;

    uploadSamplesBatch(qbatch);

    // hipStreamSynchronize(computeStream[otherStream]);

    calculateTargets();
    forwardPass();
    backwardPass();
    updateAdamParams();
    updateWeights();

    curStream = 1 - curStream;
    otherStream = 1 - otherStream;
  }

private:
  void uploadSamplesBatch(const QBatch &qbatch) {
    assert(qbatch.initialStates.rows <= d_samplesBatch[curStream].maxBatchSize);
    assert(qbatch.initialStates.rows == qbatch.batchSize);
    assert(qbatch.initialStates.rows == qbatch.successorStates.rows);
    assert(qbatch.initialStates.cols == qbatch.successorStates.cols);
    assert(qbatch.initialStates.cols == d_samplesBatch[curStream].inputDim);

    d_samplesBatch[curStream].batchSize = qbatch.batchSize;
    d_samplesBatch[curStream].futureRewardDiscount = qbatch.futureRewardDiscount;

    hipError_t err = hipMemcpy2DAsync(
        d_samplesBatch[curStream].input, d_samplesBatch[curStream].ipitch, // dst
        qbatch.initialStates.data, qbatch.initialStates.cols * sizeof(float), // src
        qbatch.initialStates.cols * sizeof(float), qbatch.initialStates.rows, // width, height
        hipMemcpyHostToDevice, computeStream[curStream]);
    CheckError(err);

    err = hipMemcpy2DAsync(
        d_samplesBatch[curStream].qinput, d_samplesBatch[curStream].qpitch, // dst
        qbatch.successorStates.data, qbatch.successorStates.cols * sizeof(float), // src
        qbatch.successorStates.cols * sizeof(float), qbatch.successorStates.rows, // width, height
        hipMemcpyHostToDevice, computeStream[curStream]);
    CheckError(err);

    err = hipMemcpyAsync(d_samplesBatch[curStream].actionIndex, qbatch.actionsTaken,
        qbatch.batchSize * sizeof(unsigned), hipMemcpyHostToDevice, computeStream[curStream]);
    CheckError(err);

    err = hipMemcpyAsync(d_samplesBatch[curStream].rewards, qbatch.rewardsGained,
        qbatch.batchSize * sizeof(float), hipMemcpyHostToDevice, computeStream[curStream]);
    CheckError(err);

    err = hipMemcpyAsync(d_samplesBatch[curStream].isTerminal, qbatch.isEndStateTerminal,
        qbatch.batchSize * sizeof(char), hipMemcpyHostToDevice, computeStream[curStream]);
    CheckError(err);
  }

  // TODO: this function can be done independently.
  void calculateTargets(void) {
    for (auto& lo : d_layerOutputs) {
      lo.batchSize = d_samplesBatch[curStream].batchSize;
    }

    // copy the batch inputs into the first layer outputs.
    hipError_t err = hipMemcpy2DAsync(
        d_layerOutputs[0].output, d_layerOutputs[0].opitch, // dst
        d_samplesBatch[curStream].qinput, d_samplesBatch[curStream].qpitch,        // src
        d_samplesBatch[curStream].inputDim * sizeof(float), d_samplesBatch[curStream].batchSize, // width, height
        hipMemcpyDeviceToDevice, computeStream[curStream]);
    CheckError(err);

    for (unsigned i = 1; i < d_layerOutputs.size(); i++) {
      LayerActivation activation = (i == d_layerOutputs.size() - 1) ?
          networkSpec.outputActivation : networkSpec.hiddenActivation;

      ForwardPassKernel::Apply(d_targetLayerWeights[i-1], d_layerOutputs[i-1], d_layerOutputs[i],
          activation, computeStream[curStream]);
    }

    LayerBatchOutputs lastLayer = d_layerOutputs[d_layerOutputs.size() - 1];
    TargetValuesKernel::Apply(lastLayer, d_samplesBatch[curStream], computeStream[curStream]);
  }

  void forwardPass(void) {
    for (auto& lo : d_layerOutputs) {
      lo.batchSize = d_samplesBatch[curStream].batchSize;
    }

    // copy the batch inputs into the first layer outputs.
    hipError_t err = hipMemcpy2DAsync(
        d_layerOutputs[0].output, d_layerOutputs[0].opitch, // dst
        d_samplesBatch[curStream].input, d_samplesBatch[curStream].ipitch,        // src
        d_samplesBatch[curStream].inputDim * sizeof(float), d_samplesBatch[curStream].batchSize, // width, height
        hipMemcpyDeviceToDevice, computeStream[curStream]);
    CheckError(err);

    for (unsigned i = 1; i < d_layerOutputs.size(); i++) {
      LayerActivation activation = (i == d_layerOutputs.size() - 1) ?
          networkSpec.outputActivation : networkSpec.hiddenActivation;

      ForwardPassKernel::Apply(d_layerWeights[i-1], d_layerOutputs[i-1], d_layerOutputs[i],
          activation, computeStream[curStream]);
    }

    LayerBatchOutputs lastLayer = d_layerOutputs[d_layerOutputs.size() - 1];
    if (networkSpec.outputActivation == LayerActivation::SOFTMAX) {
      SoftmaxKernel::Apply(lastLayer, computeStream[curStream]);
    }
  }

  void backwardPass(void) {
    generateLayerDeltas();
    generateGradient();
  }

  void generateLayerDeltas(void) {
    for (auto& ld : d_layerDeltas) {
      ld.batchSize = d_samplesBatch[curStream].batchSize;
    }

    LayerBatchDeltas lastLayerDeltas = d_layerDeltas[d_layerDeltas.size() - 1];
    LayerBatchOutputs networkOutput = d_layerOutputs[d_layerOutputs.size() - 1];

    int bpgX = (lastLayerDeltas.layerSize + TPB_X - 1) / TPB_X;
    int bpgY = (lastLayerDeltas.batchSize + TPB_Y - 1) / TPB_Y;

    lastLayerDeltasKernel<<<dim3(bpgX, bpgY, 1), dim3(TPB_X, TPB_Y, 1), 0, computeStream[curStream]>>>(
        networkOutput, d_samplesBatch[curStream], lastLayerDeltas);

    for (int i = d_layerDeltas.size() - 2; i >= 0; i--) {
      LayerWeights transposedWeights;
      transposedWeights.inputSize = d_layerWeights[i + 1].layerSize;
      transposedWeights.layerSize = d_layerWeights[i + 1].inputSize;
      transposedWeights.weights = d_transposeScratch.weights;
      transposedWeights.pitch = d_transposeScratch.pitch;

      TransposeKernel::Apply(d_layerWeights[i + 1], transposedWeights, computeStream[curStream]);

      BackwardDeltaKernel::Apply(d_layerDeltas[i + 1], transposedWeights, d_layerOutputs[i+1],
                                 d_layerDeltas[i], computeStream[curStream]);
    }
  }

  void generateGradient(void) {
    for (unsigned i = 0; i < d_layerWeights.size(); i++) {
      GradientKernel::Apply(d_layerDeltas[i], d_layerOutputs[i], d_layerGradients[i], computeStream[curStream]);
    }
  }

  void updateAdamParams(void) {
    for (unsigned i = 0; i < d_layerGradients.size(); i++) {
      int bpgX = (d_layerGradients[i].inputSize + TPB_X - 1) / TPB_X;
      int bpgY = (d_layerGradients[i].layerSize + TPB_Y - 1) / TPB_Y;

      updateMomentumAndRMS<<<dim3(bpgX, bpgY, 1), dim3(TPB_X, TPB_Y, 1), 0, computeStream[curStream]>>>(
          d_layerGradients[i], d_adamMomentum[i], d_adamRMS[i], adamBeta1, adamBeta2);
    }
  }

  void updateWeights(void) {
    for (unsigned i = 0; i < d_layerWeights.size(); i++) {
      int bpgX = (d_layerWeights[i].inputSize + TPB_X - 1) / TPB_X;
      int bpgY = (d_layerWeights[i].layerSize + TPB_Y - 1) / TPB_Y;

      updateWeightsWithAdam<<<dim3(bpgX, bpgY, 1), dim3(TPB_X, TPB_Y, 1), 0, computeStream[curStream]>>>(
          d_layerWeights[i], d_adamMomentum[i], d_adamRMS[i],
          adamBeta1, adamBeta2, adamLearnRate, adamEpsilon);
    }
  }

  void initialiseADAM(void) {
    assert(d_adamRMS.size() == d_adamMomentum.size());

    for (unsigned i = 0; i < d_adamRMS.size(); i++) {
      int bpgX = (d_adamRMS[i].inputSize + TPB_X - 1) / TPB_X;
      int bpgY = (d_adamRMS[i].layerSize + TPB_Y - 1) / TPB_Y;

      initialiseAdamWeights<<<dim3(bpgX, bpgY, 1), dim3(TPB_X, TPB_Y, 1)>>>(
          d_adamMomentum[i], d_adamRMS[i]);
    }
  }

  void initialiseOutputs(void) {
    // We initialise the outputs array for each layer to have a 1.0 at the end so that it can
    // be used as the bias input for the next layer.
    for (auto& lo : d_layerOutputs) {
      int bpgX = (lo.maxBatchSize + TPB_X - 1) / TPB_X;
      initialiseLayerOutputs<<<bpgX, TPB_X>>>(lo);
    }
  }

  void initialiseWeights(void) {
    for (auto& lw : d_layerWeights) {
      // Blocks per grid in X and Y dimensions.
      int bpgX = (lw.inputSize + TPB_X - 1) / TPB_X;
      int bpgY = (lw.layerSize + TPB_Y - 1) / TPB_Y;

      float initRange = 1.0f / sqrtf(lw.inputSize);
      initialiseLayerWeights<<<dim3(bpgX, bpgY, 1), dim3(TPB_X, TPB_Y, 1)>>>(lw, initRange, rnd);
    }

    updateTargetWeights();
  }

  void updateTargetWeights(void) {
    assert(d_layerWeights.size() == d_targetLayerWeights.size());
    for (unsigned i = 0; i < d_layerWeights.size(); i++) {
      hipError_t err = hipMemcpy2D(
          d_targetLayerWeights[i].weights, d_targetLayerWeights[i].pitch,
          d_layerWeights[i].weights, d_layerWeights[i].pitch,
          d_layerWeights[i].inputSize * sizeof(float), d_layerWeights[i].layerSize,
          hipMemcpyDeviceToDevice);

      CheckError(err);
    }
  }

  // Pre-allocated all of the device memory we will need. We should never have to malloc device
  // memory after this function is called.
  void allocDeviceMemory(void) {
    vector<unsigned> layerSizes(networkSpec.hiddenLayers.size() + 1);
    for (unsigned i = 0; i < networkSpec.hiddenLayers.size(); i++) {
      layerSizes[i] = networkSpec.hiddenLayers[i];
    }
    layerSizes[networkSpec.hiddenLayers.size()] = networkSpec.numOutputs;

    // This is for the input layer
    d_layerOutputs.push_back(
        util::NewLayerBatchOutputs(networkSpec.maxBatchSize, networkSpec.numInputs + 1));

    unsigned maxInputSize = 0;
    unsigned maxLayerSize = 0;

    for (unsigned i = 0; i < layerSizes.size(); i++) {
      unsigned prevLayerSize = i == 0 ? networkSpec.numInputs : layerSizes[i-1];

      maxInputSize = max(maxInputSize, prevLayerSize + 1);
      maxLayerSize = max(maxLayerSize, layerSizes[i]);

      d_layerWeights.push_back(util::NewLayerWeights(prevLayerSize + 1, layerSizes[i]));
      d_targetLayerWeights.push_back(util::NewLayerWeights(prevLayerSize + 1, layerSizes[i]));
      d_layerGradients.push_back(util::NewLayerWeights(prevLayerSize + 1, layerSizes[i]));
      d_layerOutputs.push_back(util::NewLayerBatchOutputs(networkSpec.maxBatchSize, layerSizes[i] + 1));
      d_layerDeltas.push_back(util::NewLayerBatchDeltas(networkSpec.maxBatchSize, layerSizes[i]));

      d_adamMomentum.push_back(util::NewLayerWeights(prevLayerSize + 1, layerSizes[i]));
      d_adamRMS.push_back(util::NewLayerWeights(prevLayerSize + 1, layerSizes[i]));
    }

    for (unsigned i = 0; i < NUM_BUFFERS; i++) {
      d_samplesBatch.push_back(
          util::NewSamplesBatch(networkSpec.maxBatchSize, networkSpec.numInputs));
    }
    d_transposeScratch = util::NewLayerWeights(maxLayerSize, maxInputSize);
  }
};


CudaNetwork::CudaNetwork(const NetworkSpec &spec) : impl(new CudaNetworkImpl(spec)) {}
CudaNetwork::~CudaNetwork() = default;

void CudaNetwork::SetWeights(const std::vector<math::MatrixView> &weights) {
    impl->SetWeights(weights);
}

void CudaNetwork::GetWeights(std::vector<math::MatrixView> &outWeights) {
  impl->GetWeights(outWeights);
}

void CudaNetwork::UpdateTarget(void) {
  impl->UpdateTarget();
}

void CudaNetwork::Train(const QBatch &qbatch) {
  impl->Train(qbatch);
}
