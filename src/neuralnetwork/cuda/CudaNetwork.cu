#include "hip/hip_runtime.h"

#include "CudaNetwork.hpp"
#include "Util.hpp"
#include "Random.hpp"
#include "SoftmaxKernel.hpp"
#include "ForwardPassKernel.hpp"
#include "TransposeKernel.hpp"
#include "BackwardDeltaKernel.hpp"
#include "GradientKernel.hpp"
#include "Constants.hpp"

#include <cassert>
#include <cmath>
#include <vector>
#include <iostream>
#include <cstdio>

#include <hiprand.h>
#include <hip/hip_runtime.h>

using namespace neuralnetwork;
using namespace neuralnetwork::cuda;
using namespace std;

// ADAM trainer parameters
static constexpr float adamBeta1 = 0.9f;
static constexpr float adamBeta2 = 0.999f;
static constexpr float adamEpsilon = 10e-8;
static constexpr float adamLearnRate = 0.001f;

static Random rnd;
static NetworkSpec networkSpec;
static vector<LayerWeights> d_layerWeights;
static vector<LayerWeights> d_layerWeightsBridge;
static vector<LayerWeights> d_layerGradients;
static vector<LayerBatchOutputs> d_layerOutputs;
static vector<LayerBatchDeltas> d_layerDeltas;
static SamplesBatch d_samplesBatch;

static LayerWeights d_transposeScratch;

// TODO: this stuff should go into a separate file. Trainer code/variables should be
// separate from network code.
static vector<LayerWeights> d_adamMomentum;
static vector<LayerWeights> d_adamRMS;

static hipStream_t uploadStream;
static hipStream_t computeStream;

static void printMatrixView(math::MatrixView view) {
  for (unsigned r = 0; r < view.rows; r++) {
    for(unsigned c = 0; c < view.cols; c++) {
      std::cout << view.data[c + r * view.cols] << " ";
    }
    std::cout << std::endl;
  }
}

static void printLayerWeights(LayerWeights d_weights) {
  math::MatrixView view;
  view.rows = d_weights.layerSize;
  view.cols = d_weights.inputSize;
  view.data = new float[view.rows * view.cols];

  hipError_t err = hipMemcpy2D(
      view.data, view.cols * sizeof(float),
      d_weights.weights, d_weights.pitch,
      view.cols * sizeof(float), view.rows,
      hipMemcpyDeviceToHost);

  CheckError(err);

  printMatrixView(view);
  delete[] view.data;
}

static void printLayerOutputs(LayerBatchOutputs d_outputs) {
  math::MatrixView view;
  view.rows = d_outputs.batchSize;
  view.cols = d_outputs.layerSize;
  view.data = new float[view.rows * view.cols];

  hipError_t err = hipMemcpy2D(
      view.data, view.cols * sizeof(float),
      d_outputs.output, d_outputs.opitch,
      view.cols * sizeof(float), view.rows,
      hipMemcpyDeviceToHost);

  CheckError(err);

  printMatrixView(view);
  delete[] view.data;
}

static void printLayerDeltas(LayerBatchDeltas d_deltas) {
  math::MatrixView view;
  view.rows = d_deltas.batchSize;
  view.cols = d_deltas.layerSize;
  view.data = new float[view.rows * view.cols];

  hipError_t err = hipMemcpy2D(
      view.data, view.cols * sizeof(float),
      d_deltas.delta, d_deltas.pitch,
      view.cols * sizeof(float), view.rows,
      hipMemcpyDeviceToHost);

  CheckError(err);

  printMatrixView(view);
  delete[] view.data;
}

// Pre-allocated all of the device memory we will need. We should never have to malloc device
// memory after this function is called.
static void allocDeviceMemory(void) {
  vector<unsigned> layerSizes(networkSpec.hiddenLayers.size() + 1);
  for (unsigned i = 0; i < networkSpec.hiddenLayers.size(); i++) {
    layerSizes[i] = networkSpec.hiddenLayers[i];
  }
  layerSizes[networkSpec.hiddenLayers.size()] = networkSpec.numOutputs;

  // This is for the input layer
  d_layerOutputs.push_back(
      util::NewLayerBatchOutputs(networkSpec.maxBatchSize, networkSpec.numInputs + 1));

  unsigned maxInputSize = 0;
  unsigned maxLayerSize = 0;

  for (unsigned i = 0; i < layerSizes.size(); i++) {
    unsigned prevLayerSize = i == 0 ? networkSpec.numInputs : layerSizes[i-1];

    maxInputSize = max(maxInputSize, prevLayerSize + 1);
    maxLayerSize = max(maxLayerSize, layerSizes[i]);

    d_layerWeights.push_back(util::NewLayerWeights(prevLayerSize + 1, layerSizes[i]));
    d_layerWeightsBridge.push_back(util::NewLayerWeights(prevLayerSize + 1, layerSizes[i]));
    d_layerGradients.push_back(util::NewLayerWeights(prevLayerSize + 1, layerSizes[i]));
    d_layerOutputs.push_back(util::NewLayerBatchOutputs(networkSpec.maxBatchSize, layerSizes[i] + 1));
    d_layerDeltas.push_back(util::NewLayerBatchDeltas(networkSpec.maxBatchSize, layerSizes[i]));

    d_adamMomentum.push_back(util::NewLayerWeights(prevLayerSize + 1, layerSizes[i]));
    d_adamRMS.push_back(util::NewLayerWeights(prevLayerSize + 1, layerSizes[i]));
  }

  d_samplesBatch =
      util::NewSamplesBatch(networkSpec.maxBatchSize, networkSpec.numInputs, networkSpec.numOutputs);

  d_transposeScratch = util::NewLayerWeights(maxLayerSize, maxInputSize);
}

static void freeDeviceMemory(void) {
  for (auto& lw : d_layerWeights) { util::DeleteLayerWeights(lw); }
  for (auto& lw : d_layerWeightsBridge) { util::DeleteLayerWeights(lw); }
  for (auto& lg : d_layerGradients) { util::DeleteLayerWeights(lg); }
  for (auto& lo : d_layerOutputs) { util::DeleteLayerBatchOutputs(lo); }
  for (auto& ld : d_layerDeltas) { util::DeleteLayerBatchDeltas(ld); }
  for (auto& am : d_adamMomentum) { util::DeleteLayerWeights(am); }
  for (auto& am : d_adamRMS) { util::DeleteLayerWeights(am); }
  util::DeleteSamplesBatch(d_samplesBatch);
  util::DeleteLayerWeights(d_transposeScratch);
}

__global__ void initialiseLayerWeights(LayerWeights layer, const float initRange, Random rnd) {
  const unsigned row = blockDim.y * blockIdx.y + threadIdx.y;
  const unsigned col = blockDim.x * blockIdx.x + threadIdx.x;

  if (row >= layer.layerSize || col >= layer.inputSize) {
    return;
  }

  float *out = layer.Elem(row, col);
  *out = initRange * (rnd.SampleUniform(col + row * layer.inputSize) * 2.0f - 1.0f);
}

static void initialiseWeights(void) {
  for (auto& lw : d_layerWeights) {
    // Blocks per grid in X and Y dimensions.
    int bpgX = (lw.inputSize + TPB_X - 1) / TPB_X;
    int bpgY = (lw.layerSize + TPB_Y - 1) / TPB_Y;

    float initRange = 1.0f / sqrtf(lw.inputSize);
    initialiseLayerWeights<<<dim3(bpgX, bpgY, 1), dim3(TPB_X, TPB_Y, 1)>>>(lw, initRange, rnd);
  }
}

__global__ void initialiseLayerOutputs(LayerBatchOutputs outputs) {
  const unsigned id = blockDim.x * blockIdx.x + threadIdx.x;
  if (id >= outputs.maxBatchSize) {
    return;
  }

  *(outputs.OutputElem(id, outputs.layerSize - 1)) = 1.0f;
}

static void initialiseOutputs(void) {
  // We initialise the outputs array for each layer to have a 1.0 at the end so that it can
  // be used as the bias input for the next layer.
  for (auto& lo : d_layerOutputs) {
    int bpgX = (lo.maxBatchSize + TPB_X - 1) / TPB_X;
    initialiseLayerOutputs<<<bpgX, TPB_X>>>(lo);
  }
}

__global__ void initialiseAdamWeights(LayerWeights momentum, LayerWeights rms) {
  assert(momentum.inputSize == rms.inputSize);
  assert(momentum.layerSize == rms.layerSize);

  const unsigned row = blockDim.y * blockIdx.y + threadIdx.y;
  const unsigned col = blockDim.x * blockIdx.x + threadIdx.x;

  if (row >= rms.layerSize || col >= rms.inputSize) {
    return;
  }

  *momentum.Elem(row, col) = 0.0f;
  *rms.Elem(row, col) = 0.0f;
}

static void initialiseADAM(void) {
  assert(d_adamRMS.size() == d_adamMomentum.size());

  for (unsigned i = 0; i < d_adamRMS.size(); i++) {
    int bpgX = (d_adamRMS[i].inputSize + TPB_X - 1) / TPB_X;
    int bpgY = (d_adamRMS[i].layerSize + TPB_Y - 1) / TPB_Y;

    initialiseAdamWeights<<<dim3(bpgX, bpgY, 1), dim3(TPB_X, TPB_Y, 1)>>>(
        d_adamMomentum[i], d_adamRMS[i]);
  }
}

void CudaNetwork::Initialise(const NetworkSpec &spec) {
  rnd = Random::Create(2048, 1337);

  uploadStream = 0;
  computeStream = 0;

  networkSpec = spec;
  assert(networkSpec.hiddenActivation != LayerActivation::SOFTMAX);

  allocDeviceMemory();
  initialiseWeights();
  initialiseOutputs();
  initialiseADAM();
}

void CudaNetwork::Cleanup(void) {
  freeDeviceMemory();
}

void CudaNetwork::SetWeights(const std::vector<math::MatrixView> &weights) {
  assert(d_layerWeights.size() == weights.size());

  for (unsigned i = 0; i < weights.size(); i++) {
    assert(weights[i].rows == d_layerWeights[i].layerSize);
    assert(weights[i].cols == d_layerWeights[i].inputSize);

    hipError_t err = hipMemcpy2D(
        d_layerWeights[i].weights, d_layerWeights[i].pitch,
        weights[i].data, weights[i].cols * sizeof(float),
        weights[i].cols * sizeof(float), weights[i].rows,
        hipMemcpyHostToDevice);

    CheckError(err);
  }
}

void CudaNetwork::GetWeights(std::vector<math::MatrixView> &outWeights) {
  assert(outWeights.size() == d_layerWeightsBridge.size());

  for (unsigned i = 0; i < outWeights.size(); i++) {
    assert(outWeights[i].rows == d_layerWeightsBridge[i].layerSize);
    assert(outWeights[i].cols == d_layerWeightsBridge[i].inputSize);

    hipError_t err = hipMemcpy2DAsync(
        outWeights[i].data, outWeights[i].cols * sizeof(float), // dst
        d_layerWeightsBridge[i].weights, d_layerWeightsBridge[i].pitch, // src
        outWeights[i].cols * sizeof(float), outWeights[i].rows, // width, height
        hipMemcpyDeviceToHost, uploadStream);

    CheckError(err);
  }
}

static void uploadSamplesBatch(const math::MatrixView &batchInputs,
                              const math::MatrixView &batchOutputs);
static void forwardPass(void);
static void backwardPass(void);
static void generateLayerDeltas(void);
static void generateGradient(void);
static void updateAdamParams(void);
static void updateWeights(void);

void CudaNetwork::Train(const math::MatrixView &batchInputs, const math::MatrixView &batchOutputs) {
    uploadSamplesBatch(batchInputs, batchOutputs);

    forwardPass();
    backwardPass();
    updateAdamParams();
    updateWeights();

    for (unsigned i = 0; i < d_layerWeights.size(); i++) {
      hipError_t err = hipMemcpy2D(
          d_layerWeightsBridge[i].weights, d_layerWeightsBridge[i].pitch,
          d_layerWeights[i].weights, d_layerWeights[i].pitch,
          d_layerWeights[i].inputSize * sizeof(float), d_layerWeights[i].layerSize,
          hipMemcpyDeviceToDevice);

      CheckError(err);
    }
}

void uploadSamplesBatch(const math::MatrixView &batchInputs, const math::MatrixView &batchOutputs) {
  assert(batchInputs.rows == batchOutputs.rows);
  assert(batchInputs.rows <= d_samplesBatch.maxBatchSize);
  assert(batchInputs.cols == d_samplesBatch.inputDim);
  assert(batchOutputs.cols == d_samplesBatch.targetOutputDim);

  d_samplesBatch.batchSize = batchInputs.rows;

  hipError_t err = hipMemcpy2D(
      d_samplesBatch.input, d_samplesBatch.ipitch, // dst
      batchInputs.data, batchInputs.cols * sizeof(float), // src
      batchInputs.cols * sizeof(float), batchInputs.rows, // width, height
      hipMemcpyHostToDevice);
  CheckError(err);

  err = hipMemcpy2D(
      d_samplesBatch.targetOutput, d_samplesBatch.opitch, // dst
      batchOutputs.data, batchOutputs.cols * sizeof(float), // src
      batchOutputs.cols * sizeof(float), batchOutputs.rows, // width, height
      hipMemcpyHostToDevice);
  CheckError(err);
}

void forwardPass(void) {
  for (auto& lo : d_layerOutputs) {
    lo.batchSize = d_samplesBatch.batchSize;
  }

  // copy the batch inputs into the first layer outputs.
  hipError_t err = hipMemcpy2DAsync(
      d_layerOutputs[0].output, d_layerOutputs[0].opitch, // dst
      d_samplesBatch.input, d_samplesBatch.ipitch,        // src
      d_samplesBatch.inputDim * sizeof(float), d_samplesBatch.batchSize, // width, height
      hipMemcpyDeviceToDevice, computeStream);
  CheckError(err);

  for (unsigned i = 1; i < d_layerOutputs.size(); i++) {
    LayerActivation activation = (i == d_layerOutputs.size() - 1) ?
        networkSpec.outputActivation : networkSpec.hiddenActivation;

    ForwardPassKernel::Apply(d_layerWeights[i-1], d_layerOutputs[i-1], d_layerOutputs[i],
        activation, rnd, networkSpec.nodeActivationRate, i == (d_layerOutputs.size() - 1),
        computeStream);

    // std::cout << "prev layer:" << std::endl;
    // printLayerOutputs(d_layerOutputs[i-1]);
    // std::cout << "layer weights:" << std::endl;
    // printLayerWeights(d_layerWeights[i-1]);
    // std::cout << "output:" << std::endl;
    // printLayerOutputs(d_layerOutputs[i]);
  }

  LayerBatchOutputs lastLayer = d_layerOutputs[d_layerOutputs.size() - 1];
  if (networkSpec.outputActivation == LayerActivation::SOFTMAX) {
    SoftmaxKernel::Apply(lastLayer, computeStream);
  }

  // math::MatrixView output = math::MatrixView::Create(lastLayer.batchSize, lastLayer.layerSize);
  //
  // err = hipMemcpy2D(
  //     output.data, output.cols * sizeof(float), // dst
  //     lastLayer.output, lastLayer.opitch, // src
  //     output.cols * sizeof(float), output.rows, // width, height
  //     hipMemcpyDeviceToHost);
  // CheckError(err);
  //
  // for (unsigned r = 0; r < output.rows; r++) {
  //   for (unsigned c = 0; c < output.cols; c++) {
  //     cout << output.data[c + r * output.cols] << "\t";
  //   }
  //   cout << endl;
  // }
  // cout << endl;
}

void backwardPass(void) {
  generateLayerDeltas();
  generateGradient();
}

__global__ void lastLayerDeltasKernel(LayerBatchOutputs networkOutput, SamplesBatch samples,
                                      LayerBatchDeltas out) {
  assert(networkOutput.layerSize == samples.targetOutputDim + 1);
  assert(out.layerSize == samples.targetOutputDim);

  const unsigned row = blockDim.y * blockIdx.y + threadIdx.y;
  const unsigned col = blockDim.x * blockIdx.x + threadIdx.x;

  if (row >= out.batchSize || col >= out.layerSize) {
    return;
  }

  // TODO: check whether reading into shared mem, doing computation, then writing to global mem
  // is faster. You never know.
  *out.Elem(row, col) = *networkOutput.OutputElem(row, col) - *samples.TargetOutputElem(row, col);
}

void generateLayerDeltas(void) {
  for (auto& ld : d_layerDeltas) {
    ld.batchSize = d_samplesBatch.batchSize;
  }

  LayerBatchDeltas lastLayerDeltas = d_layerDeltas[d_layerDeltas.size() - 1];
  LayerBatchOutputs networkOutput = d_layerOutputs[d_layerOutputs.size() - 1];

  int bpgX = (lastLayerDeltas.layerSize + TPB_X - 1) / TPB_X;
  int bpgY = (lastLayerDeltas.batchSize + TPB_Y - 1) / TPB_Y;

  lastLayerDeltasKernel<<<dim3(bpgX, bpgY, 1), dim3(TPB_X, TPB_Y, 1)>>>(
      networkOutput, d_samplesBatch, lastLayerDeltas);

  // std::cout << "layer deltas:" << std::endl;
  // printLayerDeltas(lastLayerDeltas);

  for (int i = d_layerDeltas.size() - 2; i >= 0; i--) {
    LayerWeights transposedWeights;
    transposedWeights.inputSize = d_layerWeights[i + 1].layerSize;
    transposedWeights.layerSize = d_layerWeights[i + 1].inputSize;
    transposedWeights.weights = d_transposeScratch.weights;
    transposedWeights.pitch = d_transposeScratch.pitch;

    TransposeKernel::Apply(d_layerWeights[i + 1], transposedWeights, computeStream);

    BackwardDeltaKernel::Apply(d_layerDeltas[i + 1], transposedWeights, d_layerOutputs[i+1],
                               d_layerDeltas[i], computeStream);

    // std::cout << "utweights: " << std::endl;
    // printLayerWeights(d_layerWeights[i + 1]);
    // std::cout << "tweights: " << std::endl;
    // printLayerWeights(transposedWeights);
    // std::cout << "layer deltas:" << std::endl;
    // printLayerDeltas(d_layerDeltas[i]);
  }
}

void generateGradient(void) {
  for (unsigned i = 0; i < d_layerWeights.size(); i++) {
    GradientKernel::Apply(d_layerDeltas[i], d_layerOutputs[i], d_layerGradients[i], computeStream);

    // std::cout << "gradient: " << std::endl;
    // printLayerWeights(d_layerGradients[i]);
  }
}

__global__ void updateMomentumAndRMS(LayerWeights gradient, LayerWeights momentum, LayerWeights rms,
                                      const float beta1, const float beta2) {
  const unsigned row = blockDim.y * blockIdx.y + threadIdx.y;
  const unsigned col = blockDim.x * blockIdx.x + threadIdx.x;

  if (row >= gradient.layerSize || col >= gradient.inputSize) {
    return;
  }

  float g = *gradient.Elem(row, col);
  float m = *momentum.Elem(row, col);
  float r = *rms.Elem(row, col);

  *momentum.Elem(row, col) = m * beta1 + g * (1.0f - beta1);
  *rms.Elem(row, col) = r * beta2 + g * g * (1.0f - beta2);
}

void updateAdamParams(void) {
  for (unsigned i = 0; i < d_layerGradients.size(); i++) {
    int bpgX = (d_layerGradients[i].inputSize + TPB_X - 1) / TPB_X;
    int bpgY = (d_layerGradients[i].layerSize + TPB_Y - 1) / TPB_Y;

    updateMomentumAndRMS<<<dim3(bpgX, bpgY, 1), dim3(TPB_X, TPB_Y, 1)>>>(
        d_layerGradients[i], d_adamMomentum[i], d_adamRMS[i], adamBeta1, adamBeta2);
  }
}

__global__ void updateWeightsWithAdam(LayerWeights weights, LayerWeights momentum, LayerWeights rms,
                                      const float beta1, const float beta2,
                                      const float lr, const float epsilon) {

  const unsigned row = blockDim.y * blockIdx.y + threadIdx.y;
  const unsigned col = blockDim.x * blockIdx.x + threadIdx.x;

  if (row >= rms.layerSize || col >= rms.inputSize) {
    return;
  }

  float mc = *momentum.Elem(row, col) / (1.0f - beta1);
  float rc = *rms.Elem(row, col) / (1.0f - beta2);

  *weights.Elem(row, col) -= lr * mc / sqrtf(rc + epsilon);
}

__global__ void updateWeightsWithGradient(LayerWeights weights, LayerWeights gradient,
                                          const float lr) {

  const unsigned row = blockDim.y * blockIdx.y + threadIdx.y;
  const unsigned col = blockDim.x * blockIdx.x + threadIdx.x;

  if (row >= weights.layerSize || col >= weights.inputSize) {
    return;
  }

  float d = *gradient.Elem(row, col);
  *weights.Elem(row, col) -= lr * d;
}

void updateWeights(void) {
  for (unsigned i = 0; i < d_layerWeights.size(); i++) {
    int bpgX = (d_layerWeights[i].inputSize + TPB_X - 1) / TPB_X;
    int bpgY = (d_layerWeights[i].layerSize + TPB_Y - 1) / TPB_Y;

    // updateWeightsWithGradient<<<dim3(bpgX, bpgY, 1), dim3(TPB_X, TPB_Y, 1)>>>(
    //     d_layerWeights[i], d_layerGradients[i], 0.001f);

    updateWeightsWithAdam<<<dim3(bpgX, bpgY, 1), dim3(TPB_X, TPB_Y, 1)>>>(
        d_layerWeights[i], d_adamMomentum[i], d_adamRMS[i],
        adamBeta1, adamBeta2, adamLearnRate, adamEpsilon);

    // std::cout << "new weights: " << std::endl;
    // printLayerWeights(d_layerWeights[i]);
  }
}
